#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// Geometry Visualization Server
// Copyright (c) 2018 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////

#include "gvs/optix/ptx/ray_types.hpp"

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

rtDeclareVariable(gvs::PerRayData, prd_current, rtPayload, );
rtBuffer<float4, 2> output_buffer;

rtDeclareVariable(float3, background_color, , );
rtDeclareVariable(float3, error_color, , );
rtDeclareVariable(float, miss_depth, , );

/**
 * @brief The starting point for ray generation
 */
RT_PROGRAM void pinhole_camera() {
    auto d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
    float3 ray_origin = eye;
    float3 ray_direction = optix::normalize(d.x * U + d.y * V + W);

    optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

    gvs::PerRayData prd;
    prd.depth = miss_depth;

    rtTrace(top_object, ray, prd);

    output_buffer[launch_index] = make_float4(prd.result, prd.depth);
}

/**
 * @brief Sets the ray data if no intersections are found.
 */
RT_PROGRAM void miss() {
    prd_current.result = background_color;
}

/**
 * @brief Sets the buffer output if an error occurs.
 */
RT_PROGRAM void exception() {
    const unsigned int code = rtGetExceptionCode();
    rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
    output_buffer[launch_index] = make_float4(error_color, 1.f);
}
