#include "hip/hip_runtime.h"
// ///////////////////////////////////////////////////////////////////////////////////////
// Geometry Visualization Server
// Copyright (c) 2018 Logan Barnes - All Rights Reserved
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
// ///////////////////////////////////////////////////////////////////////////////////////
// This define shouldn't be here. TODO: Figure out how to include the header properly
#define __CUDA_INCLUDE_COMPILER_INTERNAL_HEADERS__
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

struct PerRayData {
    float3 result;
    float depth;
};

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

rtDeclareVariable(PerRayData, prd_current, rtPayload, );
rtBuffer<float4, 2> output_buffer;

rtDeclareVariable(float3, background_color, , );
rtDeclareVariable(float3, error_color, , );
rtDeclareVariable(float, miss_depth, , );

RT_PROGRAM void pinhole_camera() {
    auto d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
    float3 ray_origin = eye;
    float3 ray_direction = optix::normalize(d.x * U + d.y * V + W);

    optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

    PerRayData prd;
    prd.depth = miss_depth;

    rtTrace(top_object, ray, prd);

    output_buffer[launch_index] = make_float4(prd.result, prd.depth);
}

RT_PROGRAM void miss() {
    prd_current.result = background_color;
}

RT_PROGRAM void exception() {
    const unsigned int code = rtGetExceptionCode();
    rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
    output_buffer[launch_index] = make_float4(error_color, 1.0);
}
